#include "hip/hip_runtime.h"
#include "iostream"

__global__ void matrix_multiplication_kernel_naive(const float* A, const float* B, float* C, int M, int N, int K) {

    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < M && y < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
        tmp += A[x * K + i] * B[i * N + y];
        }
        C[x * K + y] = tmp;
    }

}