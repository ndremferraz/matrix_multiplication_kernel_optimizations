#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel_coalesced(const float* A, const float* B, float* C, int M, int N, int K) {

    const uint cRow = blockIdx.x * BLOCKSIZE + (threadIdx.x /BLOCKSIZE);
    const uint cColumn = blockIdx.y * BLOCKSIZE + (threadIdx.y % BLOCKSIZE);

    // if statement is necessary to make things work under tile quantization
    if (cRow < M && cColumn < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
        tmp += A[cRow * K + i] * B[i * N + cColumn];
        }
        C[cRow * K + cColumn] = tmp;
    }

}