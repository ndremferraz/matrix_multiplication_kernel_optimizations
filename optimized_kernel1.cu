#include <iostream>
#include <hip/hip_runtime.h>

#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))

#define BLOCKSIZE 1024

__global__ void matrix_multiplication_kernel_coalesced(const float* A, const float* B, float* C, int M, int N, int K) {

    const uint cRow = blockIdx.x * BLOCKSIZE + (threadIdx.x /BLOCKSIZE);
    const uint cColumn = blockIdx.y * BLOCKSIZE + (threadIdx.y % BLOCKSIZE);

    // if statement is necessary to make things work under tile quantization
    if (cRow < M && cColumn < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
        tmp += A[cRow * K + i] * B[i * N + cColumn];
        }
        C[cRow * K + cColumn] = tmp;
    }

}


__global__ void matrix_multiplication_kernel_naive(const float* A, const float* B, float* C, int M, int N, int K) {

    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    // if statement is necessary to make things work under tile quantization
    if (x < M && y < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
        tmp += A[x * K + i] * B[i * N + y];
        }
        C[x * K + y] = tmp;
    }

}

int main() {    
    const int N = 1024;
    const int M = 1024;
    const int K = 1024;
    float *A, *B, *C; 

    hipMallocManaged(&A, M * N * sizeof(float));
    hipMallocManaged(&B, N * K * sizeof(float));
    hipMallocManaged(&C, M * K * sizeof(float));

    //I know this is not going to work as initially inteded for floats, but I just want some random values
    memset(A, 1, sizeof(A));
    memset(B, 2, sizeof(B));
    memset(C, 0, sizeof(C));   
    

    dim3 gridDim(CEIL_DIV(M,32), CEIL_DIV(N,32), 1);
    dim3 blockDim(32,32,1);

    // Launch non-coalesced kernel
    matrix_multiplication_kernel_naive<<<gridDim, blockDim>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();

    memset(C, 0, sizeof(C));

    matrix_multiplication_kernel_coalesced<<<gridDim, blockDim>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();


    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}